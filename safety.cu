#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "safety.h"

void _cuda_check(hipError_t cs, const char *file, long line)
{
    const char *errstr;

    if (cs != hipSuccess) {
        errstr = hipGetErrorString(cs);
        printf("CUDA error %s at %s:%ld.\n", errstr, file, line);
        exit(1);
    }
}
void _cublas_check(int cs, const char *file, long line)
{
    const char *errstr;

    if ((hipblasStatus_t)cs != HIPBLAS_STATUS_SUCCESS) {
        switch((hipblasStatus_t)cs) {
            case HIPBLAS_STATUS_SUCCESS:
                errstr = "HIPBLAS_STATUS_SUCCESS"; break;
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                errstr = "HIPBLAS_STATUS_NOT_INITIALIZED"; break;
            case HIPBLAS_STATUS_ALLOC_FAILED:
                errstr = "HIPBLAS_STATUS_ALLOC_FAILED"; break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                errstr = "HIPBLAS_STATUS_INVALID_VALUE"; break;
            case HIPBLAS_STATUS_ARCH_MISMATCH:
                errstr = "HIPBLAS_STATUS_ARCH_MISMATCH"; break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                errstr = "HIPBLAS_STATUS_MAPPING_ERROR"; break;
            case HIPBLAS_STATUS_EXECUTION_FAILED:
                errstr = "HIPBLAS_STATUS_EXECUTION_FAILED"; break;
            case HIPBLAS_STATUS_INTERNAL_ERROR:
                errstr = "HIPBLAS_STATUS_INTERNAL_ERROR"; break;
            default:
                errstr = "unknown";
        }
        printf("CUBLAS error %s at %s:%ld.\n", errstr, file, line);
        exit(1);
    }
}
