#include "hip/hip_runtime.h"
#include <cblas.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "dstedc.h"

void dlaed3_cpu(long NCORE, long K, double *D, double *QHAT, long LDQHAT,
    double RHO, double *DLAMDA, double *W, double *S)
// stably computes the eigendecomposition Q * diag(lambda) * Q**T  of
// diag(delta) + RHO * z * z**T  by solving an inverse eigenvalue problem.
{
    double *tau  = &S[0];
    double *orig = &S[K];
    double *v    = &S[2 * K];

    long i, j;
    double temp;

    omp_set_num_threads(NCORE);
    
    #pragma omp parallel for default(none) \
        private(i) firstprivate(K, RHO) shared(DLAMDA, W, tau, orig)
    for (i = 0; i < K; i++)
        dlaed4_cpu(K, i, DLAMDA, W, RHO, &tau[i], &orig[i]);

    // inverse eigenvalue problem: find v such that lambda(1), ..., lambda(n)
    // are exact eigenvalues of the matrix D + v * v**T.
    #pragma omp parallel for default(none) \
        private(i, j) firstprivate(K) shared(DLAMDA, W, v, tau, orig)
    for (i = 0; i < K; i++) {
        v[i] = orig[i] - DLAMDA[i] + tau[i];
        for (j = 0; j < i; j++)
            v[i] *= ((DLAMDA[i] - orig[j] - tau[j]) / (DLAMDA[i] - DLAMDA[j]));
        for (j = i+1; j < K; j++)
            v[i] *= ((orig[j] - DLAMDA[i] + tau[j]) / (DLAMDA[j] - DLAMDA[i]));
        v[i] = copysign(sqrt(v[i]), W[i]);
    }

    // compute the eigenvectors of D + v * v**T
    #pragma omp parallel for default(none) \
        private(i, j, temp) firstprivate(K, LDQHAT) \
        shared(D, DLAMDA, QHAT, v, tau, orig)
    for (j = 0; j < K; j++) {
        D[j] = tau[j] + orig[j];
        for (i = 0; i < K; i++)
            QHAT[i + j * LDQHAT] = v[i] / (orig[j] - DLAMDA[i] + tau[j]);
        temp = cblas_dnrm2(K, &QHAT[j * LDQHAT], 1); 
        for (i = 0; i < K; i++)
            QHAT[i + j * LDQHAT] /= temp;
    }
}
