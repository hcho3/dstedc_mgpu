#include "hip/hip_runtime.h"
#include <cblas.h>
#include <stdio.h>
#include <math.h>
#include <omp.h>
#include "dstedc.h"
#include "nvtx.h"

void dlaed3_cpu(long NCORE, long K, double *D, double *QHAT, long LDQHAT,
    double RHO, double *DLAMDA, double *W, double *S)
// stably computes the eigendecomposition Q * diag(lambda) * Q**T  of
// diag(delta) + RHO * z * z**T  by solving an inverse eigenvalue problem.
{
    RANGE_START("dlaed3_cpu", 1, 3);

    double *tau  = &S[0];
    double *orig = &S[K];
    double *v    = &S[2 * K];

    long i, j;
    double temp;

    omp_set_num_threads((int)NCORE);
    
    #pragma omp parallel default(none) \
        private(i, j, temp) firstprivate(K, RHO, LDQHAT) \
        shared(D, DLAMDA, QHAT, W, v, tau, orig)
    {
        #pragma omp for
        for (i = 0; i < K; i++)
            dlaed4_cpu(K, i, DLAMDA, W, RHO, &tau[i], &orig[i]);

        // inverse eigenvalue problem: find v such that lambda(1), ...,
        // lambda(n) are exact eigenvalues of the matrix D + v * v**T.
        #pragma omp for
        for (i = 0; i < K; i++) {
            temp = orig[i] - DLAMDA[i] + tau[i];
            for (j = 0; j < i; j++)
                temp *= ((DLAMDA[i] - orig[j] - tau[j])
                         / (DLAMDA[i] - DLAMDA[j]));
            for (j = i+1; j < K; j++)
                temp *= ((orig[j] - DLAMDA[i] + tau[j])
                         / (DLAMDA[j] - DLAMDA[i]));
            temp = copysign(sqrt(temp), W[i]);
            v[i] = temp;
        }

        // compute the eigenvectors of D + v * v**T
        #pragma omp for
        for (j = 0; j < K; j++) {
            D[j] = tau[j] + orig[j];
            for (i = 0; i < K; i++)
                QHAT[i + j * LDQHAT] = v[i] / (orig[j] - DLAMDA[i] + tau[j]);
            temp = cblas_dnrm2(K, &QHAT[j * LDQHAT], 1); 
            for (i = 0; i < K; i++)
                QHAT[i + j * LDQHAT] /= temp;
        }
    }

    RANGE_END(1);
}
