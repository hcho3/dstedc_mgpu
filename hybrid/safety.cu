#include <stdio.h>
#include <stdlib.h>
#include <hipblas.h>
#include "safety.h"

void _cuda_check(hipError_t cs, long line)
{
    const char *errstr;

    if (cs != hipSuccess) {
        errstr = hipGetErrorString(cs);
        printf("CUDA error %s at %ld.\n", errstr, line);
        exit(1);
    }
}
void _cublas_check(hipblasStatus_t cs, long line)
{
    const char *errstr;

    if (cs != HIPBLAS_STATUS_SUCCESS) {
        switch(cs) {
            case HIPBLAS_STATUS_SUCCESS:
                errstr = "HIPBLAS_STATUS_SUCCESS"; break;
            case HIPBLAS_STATUS_NOT_INITIALIZED:
                errstr = "HIPBLAS_STATUS_NOT_INITIALIZED"; break;
            case HIPBLAS_STATUS_ALLOC_FAILED:
                errstr = "HIPBLAS_STATUS_ALLOC_FAILED"; break;
            case HIPBLAS_STATUS_INVALID_VALUE:
                errstr = "HIPBLAS_STATUS_INVALID_VALUE"; break;
            case HIPBLAS_STATUS_ARCH_MISMATCH:
                errstr = "HIPBLAS_STATUS_ARCH_MISMATCH"; break;
            case HIPBLAS_STATUS_MAPPING_ERROR:
                errstr = "HIPBLAS_STATUS_MAPPING_ERROR"; break;
            case HIPBLAS_STATUS_EXECUTION_FAILED:
                errstr = "HIPBLAS_STATUS_EXECUTION_FAILED"; break;
            case HIPBLAS_STATUS_INTERNAL_ERROR:
                errstr = "HIPBLAS_STATUS_INTERNAL_ERROR"; break;
            default:
                errstr = "unknown";
        }
        printf("CUBLAS error %s at %ld.\n", errstr, line);
        exit(1);
    }
}
