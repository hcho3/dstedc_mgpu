#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "dstedc.h"

extern double *read_mat(const char *filename, const char *varname,
    size_t *dims);
extern void write_mat(const char *filename, const char *varname,
    double *array, size_t *dims);

double GetTimerValue(timeval time_1, timeval time_2);

int main(int argc, char **argv)
{
    const char *fin  = argv[1];
    const char *fout1 = argv[2];
    const char *fout2 = argv[3];
	timeval timer1, timer2;
    
    size_t D_dims[2], E_dims[2], Q_dims[2];
    double *D = read_mat(fin, "D", D_dims);
    double *E = read_mat(fin, "E", E_dims);
    double *Q;
    int N = (D_dims[0] > D_dims[1]) ? (int)D_dims[0] : (int)D_dims[1];
    double *D_dev, *E_dev, *Q_dev;
    double *WORK_dev;
    int *IWORK_dev;

    Q_dims[0] = Q_dims[1] = (size_t)N;
    Q = (double *)malloc(N * N * sizeof(double));
    hipMalloc((void **)&Q_dev, N * N * sizeof(double));
    hipMalloc((void **)&D_dev, N * sizeof(double));
    hipMalloc((void **)&E_dev, (N-1) * sizeof(double));
    hipMalloc((void **)&WORK_dev, (2 * N + 2 * N * N) * sizeof(double));
    hipMalloc((void **)&IWORK_dev, (3 + 5 * N) * sizeof(int));

	gettimeofday(&timer1, NULL);
    dlaed0_bootstrap(N, D, D_dev, E, E_dev, Q, Q_dev, N, WORK_dev, IWORK_dev);
	gettimeofday(&timer2, NULL);
	printf("Time: %.3lf s\n", GetTimerValue(timer1, timer2) / 1000.0 );

    write_mat(fout1, "D", D, D_dims);
    write_mat(fout2, "Q", Q, Q_dims);

    hipFree(Q_dev);
    hipFree(D_dev);
    hipFree(E_dev);
    hipFree(WORK_dev);
    hipFree(IWORK_dev);
    free(D);
    free(E);
    free(Q);

    return 0;
}

double GetTimerValue(timeval time_1, timeval time_2)
{
    int sec, usec;
    sec  = time_2.tv_sec  - time_1.tv_sec;
    usec = time_2.tv_usec - time_1.tv_usec;
    return (1000.*(double)(sec) + (double)(usec) * 0.001);
}
