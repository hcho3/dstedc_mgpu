#include "hip/hip_runtime.h"
__device__ __forceinline__ void dlaed3(hipblasHandle_t cb_handle, int K,
    double *D, double *QHAT, int LDQHAT, double RHO, double *DLAMDA,
    double *W, double *S)
// stably computes the eigendecomposition Q * diag(lambda) * Q**T  of
// diag(delta) + RHO * z * z**T  by solving an inverse eigenvalue problem.
{
    double *tau  = &S[0];
    double *orig = &S[K];
    double *v    = &S[2 * K];

    int i, j;
    double *temp = (double *)malloc(sizeof(double));

    dlaed4<<<(K+TPB-1)/TPB, TPB>>>(K, DLAMDA, W, RHO, tau, orig);
    hipDeviceSynchronize();

    // inverse eigenvalue problem: find v such that lambda(1), ..., lambda(n)
    // are exact eigenvalues of the matrix D + v * v**T.
    for (i = 0; i < K; i++) {
        v[i] = orig[i] - DLAMDA[i] + tau[i];
        for (j = 0; j < i; j++)
            v[i] *= ((DLAMDA[i] - orig[j] - tau[j]) / (DLAMDA[i] - DLAMDA[j]));
        for (j = i+1; j < K; j++)
            v[i] *= ((orig[j] - DLAMDA[i] + tau[j]) / (DLAMDA[j] - DLAMDA[i]));
        v[i] = copysign(sqrt(v[i]), W[i]);
    }

    // compute the eigenvectors of D + v * v**T
    for (i = 0; i < K; i++)
        D[i] = tau[i] + orig[i];
    for (j = 0; j < K; j++) {
        for (i = 0; i < K; i++)
            QHAT[i + j * LDQHAT] = v[i] / (orig[j] - DLAMDA[i] + tau[j]);
        hipblasDnrm2(cb_handle, K, &QHAT[j * LDQHAT], 1, temp);
        for (i = 0; i < K; i++)
            QHAT[i + j * LDQHAT] /= *temp;
    }
    free(temp);
}
